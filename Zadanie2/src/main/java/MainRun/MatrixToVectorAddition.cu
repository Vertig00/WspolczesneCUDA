
#include <hip/hip_runtime.h>
extern "C"
__global__ void add(int n, float *a, float *sum)
{
    int i = threadIdx.x;
    if (i<n)
    {
        for (int j = 0; j < n; j++)
        {
            sum[i] = sum[i] + a[i*n + j];
        }
    }

}