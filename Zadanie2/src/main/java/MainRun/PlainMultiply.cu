
#include <hip/hip_runtime.h>
extern "C"
__global__ void multiply(int n, float *a, float *b, float *sum)
{
    int i = threadIdx.x ;
    int j = threadIdx.x % n;

    if (j<n)
    {
        sum[i] = a[i] * b[j];
    }

}